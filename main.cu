#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "complex.cuh"
#include "bitmap_img.cuh"

#define LOG_INPUT if(0)
#define LOG_OUTPUT if(1)
#define LOG if(0)
#define DIM1 1024
#define DIM2 1024

__device__ int julia ( int x , int y ) {
    const float scale = 1.5;
    float jx = scale * (float)( DIM1 /2 - x) /( DIM1 /2) ;
    float jy = scale * (float)( DIM2 /2 - y) /( DIM2 /2) ;
    complex c ( -0.8 ,0.154) ;
    complex a(jx , jy);
    int i = 0;
    for (i =0; i <200; i ++) {
        a = a* a + c;
        if (a.magnitude() > 1000)
            return 0; // return 0 if (x , y) is not in set
    }
    return 1; // return 1 if (x , y) is in set
}

__global__ void kernel(unsigned char *ptr) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    unsigned int offset = x + y * gridDim.x;

    int juliaVal = julia(x, y);
    ptr[offset*3 + 2] = 255 * juliaVal;
    ptr[offset*3 + 1] = 0;
    ptr[offset*3 + 0] = 0;
}

int main()
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    unsigned char *dev_bitmap;

    hipMalloc((void**)&dev_bitmap, DIM1 * DIM2 * 3);

    dim3 grid(DIM1, DIM2);

    kernel<<<grid, 1>>>(dev_bitmap);

    auto *h_bitmap = (unsigned char*) malloc(DIM1 * DIM2 * 3);

    hipMemcpy(h_bitmap, dev_bitmap, DIM1 * DIM2 * 3, hipMemcpyDeviceToHost);

    hipFree(dev_bitmap);

    bitmap_img image(DIM2, DIM1, &h_bitmap);

    image.write();

    free(h_bitmap);

    return 0;
}
